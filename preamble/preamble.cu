#include "hip/hip_runtime.h"

#define MINMAX_TYPE int
#define sampler_t int

#define FLT_MIN          1.19209e-07
#define FLT_MAX	         1e+37

#define MAX_ARRAY_SIZE 1000

#define uchar unsigned char
#define ushort unsigned short
#define uint unsigned int
#define ulong unsigned long

__device__ inline int2 operator+(int2 a, int2 b)
{
    return make_int2(a.x + b.x, a.y + b.y);
}

__device__ inline int4 operator+(int4 a, int4 b)
{
    return make_int4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__device__ inline int2 operator*(int b, int2 a)
{
    return make_int2(b * a.x, b * a.y);
}

__device__ inline int4 operator*(int b, int4 a)
{
    return make_int4(b * a.x, b * a.y, b * a.z, b * a.w);
}

__device__ inline float pow(float x, int y) {
    return pow(float(x), float(y));
}

__device__ inline float2 sqrt(float2 a) {
    return make_float2(sqrt(a.x), sqrt(a.y));
}

__device__ inline float4 cross(float4 a, float4 b)
{ 
    return make_float4(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x, 0); 
}

__device__ inline float dot(float4 a, float4 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
}

__device__ inline float length(float4 v)
{
    return sqrtf(dot(v, v));
}

__device__ inline unsigned int atomic_add(unsigned int* address, unsigned int value) {
    return atomicAdd(address, value);
}


__device__ inline uchar clij_convert_uchar_sat(float value) {
    if (value > 255) {
        return 255;
    }
    if (value < 0) {
        return 0;
    }
    return (uchar)value;
}

__device__ inline char clij_convert_char_sat(float value) {
    if (value > 127) {
        return 127;
    }
    if (value < -128) {
        return -128;
    }
    return (char)value;
}

__device__ inline ushort clij_convert_ushort_sat(float value) {
    if (value > 65535) {
        return 65535;
    }
    if (value < 0) {
        return 0;
    }
    return (ushort)value;
}

__device__ inline short clij_convert_short_sat(float value) {
    if (value > 32767) {
        return 32767;
    }
    if (value < -32768) {
        return -32768;
    }
    return (short)value;
}

__device__ inline uint clij_convert_uint_sat(float value) {
    if (value > 4294967295) {
        return 4294967295;
    }
    if (value < 0) {
        return 0;
    }
    return (uint)value;
}

__device__ inline uint convert_uint_sat(float value) {
    if (value > 4294967295) {
        return 4294967295;
    }
    if (value < 0) {
        return 0;
    }
    return (uint)value;
}

__device__ inline int clij_convert_int_sat(float value) {
    if (value > 2147483647) {
        return 2147483647;
    }
    if (value < -2147483648) {
        return -2147483648;
    }
    return (int)value;
}

__device__ inline uint clij_convert_ulong_sat(float value) {
    if (value > 18446744073709551615) {
        return 18446744073709551615;
    }
    if (value < 0) {
        return 0;
    }
    return (ulong)value;
}

__device__ inline int clij_convert_long_sat(float value) {
    if (value > 9223372036854775807) {
        return 9223372036854775807;
    }
    if (value < -9223372036854775808 ) {
        return -9223372036854775808 ;
    }
    return (long)value;
}

__device__ inline float clij_convert_float_sat(float value) {
    return value;
}

#define get_global_size(dim) global_size_ ## dim ## _size

#define READ_IMAGE(a,b,c) READ_ ## a ## _IMAGE(a,b,c)
#define WRITE_IMAGE(a,b,c) WRITE_ ## a ## _IMAGE(a,b,c)

#define GET_IMAGE_WIDTH(image_key) IMAGE_SIZE_ ## image_key ## _WIDTH
#define GET_IMAGE_HEIGHT(image_key) IMAGE_SIZE_ ## image_key ## _HEIGHT
#define GET_IMAGE_DEPTH(image_key) IMAGE_SIZE_ ## image_key ## _DEPTH

#define CLK_NORMALIZED_COORDS_FALSE 1
#define CLK_ADDRESS_CLAMP_TO_EDGE 2
#define CLK_FILTER_NEAREST 4
#define CLK_NORMALIZED_COORDS_TRUE 8
#define CLK_ADDRESS_CLAMP 16
#define CLK_FILTER_LINEAR 32
#define CLK_ADDRESS_NONE 64

__device__ inline float2 read_buffer3df(int read_buffer_width, int read_buffer_height, int read_buffer_depth, float * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_float2(0, 0);
    }
    return make_float2(buffer_var[pos_in_buffer],0);
}

__device__ inline double2 read_buffer3dd(int read_buffer_width, int read_buffer_height, int read_buffer_depth, double * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_double2(0, 0);
    }
    return make_double2(buffer_var[pos_in_buffer],0);
}

__device__ inline char2 read_buffer3dc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, char * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_char2(0, 0);
    }
    return make_char2(buffer_var[pos_in_buffer],0);
}

__device__ inline uchar2 read_buffer3duc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uchar * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_uchar2(0, 0);
    }
    return make_uchar2(buffer_var[pos_in_buffer],0);
}

__device__ inline short2 read_buffer3ds(int read_buffer_width, int read_buffer_height, int read_buffer_depth, short * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_short2(0, 0);
    }
    return make_short2(buffer_var[pos_in_buffer],0);
}

__device__ inline ushort2 read_buffer3dus(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ushort * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_ushort2(0, 0);
    }
    return make_ushort2(buffer_var[pos_in_buffer],0);
}

__device__ inline int2 read_buffer3di(int read_buffer_width, int read_buffer_height, int read_buffer_depth, int * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_int2(0, 0);
    }
    return make_int2(buffer_var[pos_in_buffer],0);
}

__device__ inline uint2 read_buffer3dui(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uint * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_uint2(0, 0);
    }
    return make_uint2(buffer_var[pos_in_buffer],0);
}

__device__ inline long2 read_buffer3dl(int read_buffer_width, int read_buffer_height, int read_buffer_depth, long * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_long2(0, 0);
    }
    return make_long2(buffer_var[pos_in_buffer],0);
}

__device__ inline ulong2 read_buffer3dul(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ulong * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_ulong2(0, 0);
    }
    return make_ulong2(buffer_var[pos_in_buffer],0);
}


__device__ inline void write_buffer3df(int write_buffer_width, int write_buffer_height, int write_buffer_depth, float * buffer_var, int4 pos, float value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dd(int write_buffer_width, int write_buffer_height, int write_buffer_depth, double * buffer_var, int4 pos, double value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, char * buffer_var, int4 pos, char value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3duc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uchar * buffer_var, int4 pos, uchar value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3ds(int write_buffer_width, int write_buffer_height, int write_buffer_depth, short * buffer_var, int4 pos, short value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dus(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ushort * buffer_var, int4 pos, ushort value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3di(int write_buffer_width, int write_buffer_height, int write_buffer_depth, int * buffer_var, int4 pos, int value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dui(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uint * buffer_var, int4 pos, uint value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dl(int write_buffer_width, int write_buffer_height, int write_buffer_depth, long * buffer_var, int4 pos, uint value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dul(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ulong * buffer_var, int4 pos, uint value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}



__device__ inline float2 read_buffer2df(int read_buffer_width, int read_buffer_height, int read_buffer_depth, float * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_float2(0, 0);
    }
    return make_float2(buffer_var[pos_in_buffer],0);
}

__device__ inline double2 read_buffer2dd(int read_buffer_width, int read_buffer_height, int read_buffer_depth, double * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_double2(0, 0);
    }
    return make_double2(buffer_var[pos_in_buffer],0);
}

__device__ inline char2 read_buffer2dc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, char * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_char2(0, 0);
    }
    return make_char2(buffer_var[pos_in_buffer],0);
}

__device__ inline uchar2 read_buffer2duc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uchar * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_uchar2(0, 0);
    }
    return make_uchar2(buffer_var[pos_in_buffer],0);
}

__device__ inline short2 read_buffer2ds(int read_buffer_width, int read_buffer_height, int read_buffer_depth, short * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_short2(0, 0);
    }
    return make_short2(buffer_var[pos_in_buffer],0);
}

__device__ inline ushort2 read_buffer2dus(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ushort * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_ushort2(0, 0);
    }
    return make_ushort2(buffer_var[pos_in_buffer],0);
}

__device__ inline int2 read_buffer2di(int read_buffer_width, int read_buffer_height, int read_buffer_depth, int * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_int2(0, 0);
    }
    return make_int2(buffer_var[pos_in_buffer],0);
}

__device__ inline uint2 read_buffer2dui(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uint * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_uint2(0, 0);
    }
    return make_uint2(buffer_var[pos_in_buffer],0);
}

__device__ inline long2 read_buffer2dl(int read_buffer_width, int read_buffer_height, int read_buffer_depth, long * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_long2(0, 0);
    }
    return make_long2(buffer_var[pos_in_buffer],0);
}

__device__ inline ulong2 read_buffer2dul(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ulong * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_ulong2(0, 0);
    }
    return make_ulong2(buffer_var[pos_in_buffer],0);
}


__device__ inline void write_buffer2df(int write_buffer_width, int write_buffer_height, int write_buffer_depth, float * buffer_var, int2 pos, float value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dd(int write_buffer_width, int write_buffer_height, int write_buffer_depth, double * buffer_var, int2 pos, double value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, char * buffer_var, int2 pos, char value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2duc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uchar * buffer_var, int2 pos, uchar value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2ds(int write_buffer_width, int write_buffer_height, int write_buffer_depth, short * buffer_var, int2 pos, short value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dus(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ushort * buffer_var, int2 pos, ushort value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2di(int write_buffer_width, int write_buffer_height, int write_buffer_depth, int * buffer_var, int2 pos, int value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dui(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uint * buffer_var, int2 pos, uint value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dl(int write_buffer_width, int write_buffer_height, int write_buffer_depth, long * buffer_var, int2 pos, long value )
{
   int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dul(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ulong * buffer_var, int2 pos, ulong value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}



// __device__ inline float2 read_buffer1df(int read_buffer_width, int read_buffer_height, int read_buffer_depth, float * buffer_var, int sampler, int position )
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_float2(0, 0);
//     }
//     return make_float2(buffer_var[pos_in_buffer],0);
// }

// __device__ inline double2 read_buffer1dd(int read_buffer_width, int read_buffer_height, int read_buffer_depth, double * buffer_var, int sampler, int position )
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_double2(0, 0);
//     }
//     return make_double2(buffer_var[pos_in_buffer],0);
// }

// __device__ inline char2 read_buffer1dc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, char * buffer_var, int sampler, int position )
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_char2(0, 0);
//     }
//     return make_char2(buffer_var[pos_in_buffer],0);
// }

// __device__ inline uchar2 read_buffer1duc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uchar * buffer_var, int sampler, int position )
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_uchar2(0, 0);
//     }
//     return make_uchar2(buffer_var[pos_in_buffer],0);
// }

// __device__ inline short2 read_buffer1ds(int read_buffer_width, int read_buffer_height, int read_buffer_depth, short * buffer_var, int sampler, int position )
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_short2(0, 0);
//     }
//     return make_short2(buffer_var[pos_in_buffer],0);
// }

// __device__ inline ushort2 read_buffer1dus(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ushort * buffer_var, int sampler, int position )
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_ushort2(0, 0);
//     }
//     return make_ushort2(buffer_var[pos_in_buffer],0);
// }

// __device__ inline int2 read_buffer1di(int read_buffer_width, int read_buffer_height, int read_buffer_depth, int * buffer_var, int sampler, int position )
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_int2(0, 0);
//     }
//     return make_int2(buffer_var[pos_in_buffer],0);
// }

// __device__ inline uint2 read_buffer1dui(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uint * buffer_var, int sampler, int position ) 
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_uint2(0, 0);
//     }
//     return make_uint2(buffer_var[pos_in_buffer],0);
// }

// __device__ inline long2 read_buffer1dl(int read_buffer_width, int read_buffer_height, int read_buffer_depth, long * buffer_var, int sampler, int position )
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_long2(0, 0);
//     }
//     return make_long2(buffer_var[pos_in_buffer],0);
// }

// __device__ inline ulong2 read_buffer1dul(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ulong * buffer_var, int sampler, int position )
// {
//     int pos = position;

//     pos = max((MINMAX_TYPE)pos, (MINMAX_TYPE)0);
//     pos = min((MINMAX_TYPE)pos, (MINMAX_TYPE)read_buffer_width - 1);

//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= read_buffer_width) {
//         return make_ulong2(0, 0);
//     }
//     return make_ulong2(buffer_var[pos_in_buffer],0);
// }


// __device__ inline void write_buffer1df(int write_buffer_width, int write_buffer_height, int write_buffer_depth, float * buffer_var, int pos, float value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }

// __device__ inline void write_buffer1dd(int write_buffer_width, int write_buffer_height, int write_buffer_depth, double * buffer_var, int pos, double value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }

// __device__ inline void write_buffer1dc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, char * buffer_var, int pos, char value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }

// __device__ inline void write_buffer1duc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uchar * buffer_var, int pos, uchar value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }

// __device__ inline void write_buffer1ds(int write_buffer_width, int write_buffer_height, int write_buffer_depth, short * buffer_var, int pos, short value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }

// __device__ inline void write_buffer1dus(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ushort * buffer_var, int pos, ushort value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }

// __device__ inline void write_buffer1di(int write_buffer_width, int write_buffer_height, int write_buffer_depth, int * buffer_var, int pos, int value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }

// __device__ inline void write_buffer1dui(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uint * buffer_var, int pos, uint value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }

// __device__ inline void write_buffer1dl(int write_buffer_width, int write_buffer_height, int write_buffer_depth, long * buffer_var, int pos, long value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }

// __device__ inline void write_buffer1dul(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ulong * buffer_var, int pos, ulong value )
// {
//     int pos_in_buffer = pos;
//     if (pos < 0 || pos >= write_buffer_width) {
//         return;
//     }
//     buffer_var[pos_in_buffer] = value;
// }
