
#include <hip/hip_runtime.h>
#define MINMAX_TYPE int
#define sampler_t int

#define FLT_MIN          1.19209e-07
#define FLT_MAX	         1e+37

#define MAX_ARRAY_SIZE 1000

#define uchar unsigned char
#define ushort unsigned short
#define uint unsigned int
#define ulong unsigned long


__device__ inline uchar clij_convert_uchar_sat(float value) {
    if (value > 255) {
        return 255;
    }
    if (value < 0) {
        return 0;
    }
    return (uchar)value;
}


__device__ inline char clij_convert_char_sat(float value) {
    if (value > 127) {
        return 127;
    }
    if (value < -128) {
        return -128;
    }
    return (char)value;
}


__device__ inline ushort clij_convert_ushort_sat(float value) {
    if (value > 65535) {
        return 65535;
    }
    if (value < 0) {
        return 0;
    }
    return (ushort)value;
}


__device__ inline short clij_convert_short_sat(float value) {
    if (value > 32767) {
        return 32767;
    }
    if (value < -32768) {
        return -32768;
    }
    return (short)value;
}

__device__ inline uint clij_convert_uint_sat(float value) {
    if (value > 4294967295) {
        return 4294967295;
    }
    if (value < 0) {
        return 0;
    }
    return (uint)value;
}

__device__ inline uint convert_uint_sat(float value) {
    if (value > 4294967295) {
        return 4294967295;
    }
    if (value < 0) {
        return 0;
    }
    return (uint)value;
}


__device__ inline int clij_convert_int_sat(float value) {
    if (value > 2147483647) {
        return 2147483647;
    }
    if (value < -2147483648) {
        return -2147483648;
    }
    return (int)value;
}


__device__ inline uint clij_convert_ulong_sat(float value) {
    if (value > 18446744073709551615) {
        return 18446744073709551615;
    }
    if (value < 0) {
        return 0;
    }
    return (ulong)value;
}

__device__ inline int clij_convert_long_sat(float value) {
    if (value > 9223372036854775807) {
        return 9223372036854775807;
    }
    if (value < -9223372036854775808 ) {
        return -9223372036854775808 ;
    }
    return (long)value;
}

__device__ inline float clij_convert_float_sat(float value) {
    return value;
}


__device__ inline float2 read_buffer3df(int read_buffer_width, int read_buffer_height, int read_buffer_depth, float * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_float2(0, 0);
    }
    return make_float2(buffer_var[pos_in_buffer],0);
}

__device__ inline double2 read_buffer3dd(int read_buffer_width, int read_buffer_height, int read_buffer_depth, double * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_double2(0, 0);
    }
    return make_double2(buffer_var[pos_in_buffer],0);
}

__device__ inline char2 read_buffer3dc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, char * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_char2(0, 0);
    }
    return make_char2(buffer_var[pos_in_buffer],0);
}

__device__ inline uchar2 read_buffer3duc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uchar * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_uchar2(0, 0);
    }
    return make_uchar2(buffer_var[pos_in_buffer],0);
}

__device__ inline short2 read_buffer3ds(int read_buffer_width, int read_buffer_height, int read_buffer_depth, short * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_short2(0, 0);
    }
    return make_short2(buffer_var[pos_in_buffer],0);
}

__device__ inline ushort2 read_buffer3dus(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ushort * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_ushort2(0, 0);
    }
    return make_ushort2(buffer_var[pos_in_buffer],0);
}

__device__ inline int2 read_buffer3di(int read_buffer_width, int read_buffer_height, int read_buffer_depth, int * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_int2(0, 0);
    }
    return make_int2(buffer_var[pos_in_buffer],0);
}

__device__ inline uint2 read_buffer3dui(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uint * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_uint2(0, 0);
    }
    return make_uint2(buffer_var[pos_in_buffer],0);
}

__device__ inline long2 read_buffer3dl(int read_buffer_width, int read_buffer_height, int read_buffer_depth, long * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_long2(0, 0);
    }
    return make_long2(buffer_var[pos_in_buffer],0);
}

__device__ inline ulong2 read_buffer3dul(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ulong * buffer_var, int sampler, int4 position )
{
    int4 pos = make_int4(position.x, position.y, position.z, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width + pos.z * read_buffer_width * read_buffer_height;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height || pos.z < 0 || pos.z >= read_buffer_depth) {
        return make_ulong2(0, 0);
    }
    return make_ulong2(buffer_var[pos_in_buffer],0);
}

__device__ inline void write_buffer3df(int write_buffer_width, int write_buffer_height, int write_buffer_depth, float * buffer_var, int4 pos, float value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dd(int write_buffer_width, int write_buffer_height, int write_buffer_depth, double * buffer_var, int4 pos, double value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, char * buffer_var, int4 pos, char value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3duc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uchar * buffer_var, int4 pos, uchar value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3ds(int write_buffer_width, int write_buffer_height, int write_buffer_depth, short * buffer_var, int4 pos, short value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dus(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ushort * buffer_var, int4 pos, ushort value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3di(int write_buffer_width, int write_buffer_height, int write_buffer_depth, int * buffer_var, int4 pos, int value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dui(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uint * buffer_var, int4 pos, uint value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dl(int write_buffer_width, int write_buffer_height, int write_buffer_depth, long * buffer_var, int4 pos, uint value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer3dul(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ulong * buffer_var, int4 pos, uint value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width + pos.z * write_buffer_width * write_buffer_height;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height || pos.z < 0 || pos.z >= write_buffer_depth) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}


__device__ inline float2 read_buffer2df(int read_buffer_width, int read_buffer_height, int read_buffer_depth, float * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_float2(0, 0);
    }
    return make_float2(buffer_var[pos_in_buffer],0);
}

__device__ inline double2 read_buffer2dd(int read_buffer_width, int read_buffer_height, int read_buffer_depth, double * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_double2(0, 0);
    }
    return make_double2(buffer_var[pos_in_buffer],0);
}

__device__ inline char2 read_buffer2dc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, char * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_char2(0, 0);
    }
    return make_char2(buffer_var[pos_in_buffer],0);
}

__device__ inline uchar2 read_buffer2duc(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uchar * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_uchar2(0, 0);
    }
    return make_uchar2(buffer_var[pos_in_buffer],0);
}

__device__ inline short2 read_buffer2ds(int read_buffer_width, int read_buffer_height, int read_buffer_depth, short * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_short2(0, 0);
    }
    return make_short2(buffer_var[pos_in_buffer],0);
}

__device__ inline ushort2 read_buffer2dus(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ushort * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_ushort2(0, 0);
    }
    return make_ushort2(buffer_var[pos_in_buffer],0);
}

__device__ inline int2 read_buffer2di(int read_buffer_width, int read_buffer_height, int read_buffer_depth, int * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_int2(0, 0);
    }
    return make_int2(buffer_var[pos_in_buffer],0);
}

__device__ inline uint2 read_buffer2dui(int read_buffer_width, int read_buffer_height, int read_buffer_depth, uint * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_uint2(0, 0);
    }
    return make_uint2(buffer_var[pos_in_buffer],0);
}

__device__ inline long2 read_buffer2dl(int read_buffer_width, int read_buffer_height, int read_buffer_depth, long * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_long2(0, 0);
    }
    return make_long2(buffer_var[pos_in_buffer],0);
}

__device__ inline ulong2 read_buffer2dul(int read_buffer_width, int read_buffer_height, int read_buffer_depth, ulong * buffer_var, int sampler, int2 position )
{
    int4 pos = make_int4(position.x, position.y, 0, 0);
    
    pos.x = max((MINMAX_TYPE)pos.x, (MINMAX_TYPE)0);
    pos.y = max((MINMAX_TYPE)pos.y, (MINMAX_TYPE)0);
    pos.z = max((MINMAX_TYPE)pos.z, (MINMAX_TYPE)0);
    pos.x = min((MINMAX_TYPE)pos.x, (MINMAX_TYPE)read_buffer_width - 1);
    pos.y = min((MINMAX_TYPE)pos.y, (MINMAX_TYPE)read_buffer_height - 1);
    pos.z = min((MINMAX_TYPE)pos.z, (MINMAX_TYPE)read_buffer_depth - 1);

    int pos_in_buffer = pos.x + pos.y * read_buffer_width;
    if (pos.x < 0 || pos.x >= read_buffer_width || pos.y < 0 || pos.y >= read_buffer_height) {
        return make_ulong2(0, 0);
    }
    return make_ulong2(buffer_var[pos_in_buffer],0);
}

__device__ inline void write_buffer2df(int write_buffer_width, int write_buffer_height, int write_buffer_depth, float * buffer_var, int2 pos, float value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dd(int write_buffer_width, int write_buffer_height, int write_buffer_depth, double * buffer_var, int2 pos, double value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, char * buffer_var, int2 pos, char value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2duc(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uchar * buffer_var, int2 pos, uchar value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2ds(int write_buffer_width, int write_buffer_height, int write_buffer_depth, short * buffer_var, int2 pos, short value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dus(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ushort * buffer_var, int2 pos, ushort value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2di(int write_buffer_width, int write_buffer_height, int write_buffer_depth, int * buffer_var, int2 pos, int value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dui(int write_buffer_width, int write_buffer_height, int write_buffer_depth, uint * buffer_var, int2 pos, uint value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dl(int write_buffer_width, int write_buffer_height, int write_buffer_depth, long * buffer_var, int2 pos, uint value )
{
   int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}

__device__ inline void write_buffer2dul(int write_buffer_width, int write_buffer_height, int write_buffer_depth, ulong * buffer_var, int2 pos, uint value )
{
    int pos_in_buffer = pos.x + pos.y * write_buffer_width;
    if (pos.x < 0 || pos.x >= write_buffer_width || pos.y < 0 || pos.y >= write_buffer_height) {
        return;
    }
    buffer_var[pos_in_buffer] = value;
}